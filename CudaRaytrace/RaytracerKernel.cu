#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <glm/vec4.hpp>
#include <glm/packing.hpp>
#include <glm/glm.hpp>
#include <hiprand/hiprand_kernel.h>

#include "Window.h"
#include "cuda_errors.h"
#include "FrameBuffer.h"
#include "Ray.h"
#include "Sphere.h"
#include "World.h"
#include "Camera.h"

#include "RaytracerKernel.h"

__global__ void create_world(Hittable** d_list, unsigned int d_list_size, Hittable** d_world, Camera** d_camera, CameraInfo camera_info) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		d_list[0] = new Sphere(glm::vec3(0, 0, -1), 0.5f, new Lambertian(glm::vec3(0.8f,0.3f,0.3f)));
		d_list[1] = new Sphere(glm::vec3(0, -100.5, -1), 100.0f, new Lambertian(glm::vec3(0.8f, 0.8f, 0.0f)));
		d_list[2] = new Sphere(glm::vec3(-1.01, 0, -1), 0.5f, new Dielectric(1.5f));
		d_list[3] = new Sphere(glm::vec3(-1, 10, -1), 0.5f, new Dielectric(1.5f));
		d_list[4] = new Sphere(glm::vec3(1, 0, -1), 0.5f, new Metal(glm::vec3(0.8f, 0.8f, 0.8f), 0.3f));
		*d_world = new World(d_list, d_list_size);
		*d_camera = camera_info.constructCamera();
	}
}

__global__ void set_camera(Camera** d_camera, glm::vec3 position, glm::vec3 forward, glm::vec3 up) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		(*d_camera)->setPosition(position);
		(*d_camera)->setRotation(forward, up);
	}
}

__global__ void free_world(Hittable** d_list, unsigned int d_list_size, Hittable** d_world, Camera** d_camera) {
	for (int i = 0; i < d_list_size; i++) {
		delete d_list[i];
	}
	delete* d_world;
	delete* d_camera;
}

__global__ void render_init(int width, int height, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) return;
	int pixel_index = j * width + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void raytrace(frameBuffer fb, Hittable** world, Camera** camera, hiprandState* rand_state) {

	// X AND Y coordinates
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	// return early if we're outside of the frame buffer
	if ((i >= fb.width) || (j >= fb.height)) return;

	int pixel_idx = j * fb.width + i;

	hiprandState local_rand_state = rand_state[pixel_idx];

	glm::vec3 col = glm::vec3(0.0f, 0.0f, 0.0f);

	// samples
	const int ns = 3;

	for (int s = 0; s < ns; s++) {
		// normalized screen coordinates
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(fb.width);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(fb.height);
		Ray r = (*camera)->getRay(u, v);
		col += fb.color(r, world, &local_rand_state);
	}
	rand_state[pixel_idx] = local_rand_state;
	col /= float(ns);
	col[0] = sqrtf(col[0]);
	col[1] = sqrtf(col[1]);
	col[2] = sqrtf(col[2]);

	fb.writePixel(i, j, glm::vec4(col, 1.0f));
}

kernelInfo::kernelInfo(hipGraphicsResource_t resources, int nx, int ny) {
	this->resources = resources;

	camera_info = CameraInfo(glm::vec3(0.0f,0.0f,0.0f), glm::vec3(0.0f,0.0f,0.0f), 90.0f, nx, ny);

	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));

	checkCudaErrors(hipMalloc((void**)&d_rand_state, nx * ny * sizeof(hiprandState)));

	list_size = 5;

	checkCudaErrors(hipMalloc((void**)&d_list, list_size * sizeof(Hittable*)));

	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hittable*)));
	create_world << <1, 1 >> > (d_list, list_size, d_world, d_camera, camera_info);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	int tx = 8;
	int ty = 8;

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render_init << <blocks, threads >> > (nx, ny, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void kernelInfo::setCamera(glm::vec3 position, glm::vec3 forward, glm::vec3 up) {
	set_camera << <1, 1 >> > (d_camera, position, forward, up);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void kernelInfo::render(int nx, int ny) {
	frameBuffer fb(nx, ny);

	checkCudaErrors(hipGraphicsMapResources(1, &resources, NULL));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&fb.device_ptr, &buffer_size, resources));

	int tx = 8;
	int ty = 8;

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	raytrace << <blocks, threads >> > (fb, d_world, d_camera, d_rand_state);
	checkCudaErrors(hipGetLastError());
	// wait for the gpu to finish
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipGraphicsUnmapResources(1, &resources, NULL));
}

//void kernelInfo::setCamera(glm::vec3 pos, glm::vec3 rotation) {
//	set_camera<<<1, 1 >>>(d_camera, pos);
//	checkCudaErrors(hipGetLastError());
//	checkCudaErrors(hipDeviceSynchronize());
//}

void kernelInfo::destroy() {

	free_world<<<1, 1>>> (d_list, list_size, d_world, d_camera);

	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_rand_state));
}
